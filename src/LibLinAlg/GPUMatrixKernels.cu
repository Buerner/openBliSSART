#include "hip/hip_runtime.h"
//
// This file is part of openBliSSART.
//
// Copyright (c) 2007-2011, Alexander Lehmann <lehmanna@in.tum.de>
//                          Felix Weninger <felix@weninger.de>
//                          Bjoern Schuller <schuller@tum.de>
//
// Institute for Human-Machine Communication
// Technische Universitaet Muenchen (TUM), D-80333 Munich, Germany
//
// openBliSSART is free software: you can redistribute it and/or modify it under
// the terms of the GNU General Public License as published by the Free Software
// Foundation, either version 2 of the License, or (at your option) any later
// version.
//
// openBliSSART is distributed in the hope that it will be useful, but WITHOUT
// ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
// details.
//
// You should have received a copy of the GNU General Public License along with
// openBliSSART.  If not, see <http://www.gnu.org/licenses/>.
//


#include <hip/hip_runtime.h>


namespace blissart {


namespace linalg {


namespace gpu {


int blocksize = 4;


__global__ void MatrixAdd_d(const double *a, const double *b, double *c, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col * rows + row;
    if(col < cols && row < rows) 
        c[index] = a[index] + b[index];
}


void apply_add(const double* a, const double* b, double* c, int m, int n) {
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(n / dimBlock.x + 1, m / dimBlock.y + 1);
    MatrixAdd_d<<<dimGrid, dimBlock>>>(a, b, c, m, n);
    hipDeviceSynchronize();
}


__global__ void MatrixSub_d(const double *a, const double *b, double *c, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col * rows + row;
    if(col < cols && row < rows) 
        c[index] = a[index] - b[index];
}


void apply_sub(const double* a, const double* b, double* c, int m, int n) {
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(n / dimBlock.x + 1, m / dimBlock.y + 1);
    MatrixSub_d<<<dimGrid, dimBlock>>>(a, b, c, m, n);
    hipDeviceSynchronize();
}


__global__ void MatrixMul_d(const double *a, const double *b, double *c, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col * rows + row;
    if(col < cols && row < rows) 
        c[index] = a[index] * b[index];
}


void apply_mul(const double* a, const double* b, double* c, int m, int n) {
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(n / dimBlock.x + 1, m / dimBlock.y + 1);
    MatrixMul_d<<<dimGrid, dimBlock>>>(a, b, c, m, n);
    hipDeviceSynchronize();
}


__global__ void MatrixDiv_d(const double *a, const double *b, double *c, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col * rows + row;
    if(col < cols && row < rows) 
        c[index] = a[index] / b[index];
}


void apply_div(const double* a, const double* b, double* c, int m, int n) {
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(n / dimBlock.x + 1, m / dimBlock.y + 1);
    MatrixDiv_d<<<dimGrid, dimBlock>>>(a, b, c, m, n);
    hipDeviceSynchronize();
}


__global__ void MatrixPow_d(const double *a, const double b, double *c, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col * rows + row;
    if(col < cols && row < rows) 
        c[index] = pow(a[index], b);
}


void apply_pow(const double* a, const double b, double* c, int m, int n) {
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(n / dimBlock.x + 1, m / dimBlock.y + 1);
    MatrixPow_d<<<dimGrid, dimBlock>>>(a, b, c, m, n);
    hipDeviceSynchronize();
}


__global__ void SetZero_d(double* a, int rows, int cols, 
                          int startRow, int startCol, int endRow, int endCol)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col * rows + row;
    if (col >= startCol && col <= endCol && row >= startRow && row <= endRow)
        a[index] = 0.0f;
}


void set_to_zero(double* a, int rows, int cols, 
                 int startRow, int startCol, int endRow, int endCol)
{
    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(cols / dimBlock.x + 1, rows / dimBlock.y + 1);
    SetZero_d<<<dimGrid, dimBlock>>>
        (a, rows, cols, startRow, startCol, endRow, endCol);
    hipDeviceSynchronize();
}


} // namespace gpu


} // namespace linalg


} // namespace blissart

